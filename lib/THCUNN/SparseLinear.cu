#include "THCUNN.h"
#include "common.h"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"

#include <hipsparse.h>
#include <thrust/device_vector.h>

static hipsparseHandle_t cusparse_handle = 0;

static void init_cusparse() {
  if (cusparse_handle == 0) {
    hipsparseStatus_t status = hipsparseCreate(&cusparse_handle);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
      THError("CUSPARSE Library initialization failed");
    }
  }
}

#ifdef CUDA_HALF_TENSOR
void THNN_CudaHalfSparseLinear_updateOutput(
          THCState *state,
          THCudaHalfTensor *input,
          THCudaHalfTensor *output,
          THCudaHalfTensor *weight,
          THCudaHalfTensor *bias) {
  THError("THCudaHalfTensor not supported with SparseLinear");
}

void THNN_CudaHalfSparseLinear_accGradParameters(
          THCState *state,
          THCudaHalfTensor *input,
          THCudaHalfTensor *gradOutput,
          THCudaHalfTensor *gradWeight,
          THCudaHalfTensor *gradBias,
          THCudaHalfTensor *weight,
          THCudaHalfTensor *bias,
          double weightDecay,
          double scale) {
  THError("THCudaHalfTensor not supported with SparseLinear");
}

void THNN_CudaHalfSparseLinear_legacyUpdateOutput(
          THCState *state,
          THCudaHalfTensor *input,
          THCudaHalfTensor *output,
          THCudaHalfTensor *weight,
          THCudaHalfTensor *bias) {
  THError("THCudaHalfTensor not supported with SparseLinear");
}

void THNN_CudaHalfSparseLinear_legacyAccGradParameters(
          THCState *state,
          THCudaHalfTensor *input,
          THCudaHalfTensor *gradOutput,
          THCudaHalfTensor *gradWeight,
          THCudaHalfTensor *gradBias,
          THCudaHalfTensor *weight,
          THCudaHalfTensor *bias,
          double weightDecay,
          double scale) {
  THError("THCudaHalfTensor not supported with SparseLinear");
}

void THNN_CudaHalfSparseLinear_zeroGradParameters(
          THCState *state,
          THCudaHalfTensor *gradWeight,
          THCudaHalfTensor *gradBias,
          THCudaHalfTensor *lastInput) {
  THError("THCudaHalfTensor not supported with SparseLinear");
}

<<<<<<< HEAD
void THNN_CudaSparseLinear_updateParameters(
=======
void THNN_CudaHalfSparseLinear_updateParameters(
>>>>>>> upstream/master
          THCState *state,
          THCudaHalfTensor *weight,
          THCudaHalfTensor *bias,
          THCudaHalfTensor *gradWeight,
          THCudaHalfTensor *gradBias,
          THCudaHalfTensor *lastInput,
          double learningRate) {
  THError("THCudaHalfTensor not supported with SparseLinear");
}
#endif

#include "generic/SparseLinear.cu"
#include "THCGenerateFloatType.h"
#include "generic/SparseLinear.cu"
#include "THCGenerateDoubleType.h"
