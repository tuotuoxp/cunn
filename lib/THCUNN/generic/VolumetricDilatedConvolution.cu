
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/VolumetricDilatedConvolution.cu"
#else

void THNN_(VolumetricDilatedConvolution_updateOutput)(
           THCState *state,
           THCTensor  *input,
           THCTensor  *output,
           THCTensor  *weight,
           THCTensor  *bias,
           THCTensor  *columns,
           THCTensor  *ones,
           int kT, int kW, int kH,
           int dT, int dW, int dH,
           int padT, int padW, int padH,
           int dilationT, int dilationW, int dilationH) {

  THCUNN_assertSameGPU(state, 5, input, output, weight, columns, ones);
  if (bias) {
    THCUNN_assertSameGPU(state, 2, weight, bias);
  }
  THCUNN_argCheck(state, input->nDimension == 4 || input->nDimension == 5, 2, input,
                  "4D or 5D (batch mode) tensor expected for input, but got: %s");
  THCUNN_argCheck(state, weight->nDimension == 5, 4, weight,
                  "5D (nOutputPlane x nInputPlane x kT x kH x kW) tensor "
                  "expected for weight, but got: %s");
  THArgCheck(!bias || weight->size[0] == bias->size[0], 4, "nOutputPlane mismatch in weight and bias");
  THArgCheck(kT > 0 && kW > 0 && kH > 0, 8, "kernel size should be greater than zero");
  THArgCheck(dT > 0 && dW > 0 && dH > 0, 10, "stride should be greater than zero");
  THArgCheck(dilationT > 0 && dilationW > 0 && dilationH > 0, 16, "dilation should be greater than 0");

  // Params:
  int nInputPlane = weight->size[1];
  int nOutputPlane = weight->size[0];

  int batch = 1;
  if (input->nDimension == 4) {
    THArgCheck(input->size[0] == nInputPlane, 2, "input channels and nInputPlane dont match");
    // Force batch
    batch = 0;
    THCTensor_(resize5d)(state, input, 1, input->size[0], input->size[1], input->size[2], input->size[3]);
  } else {
    THArgCheck(input->size[1] == nInputPlane, 2, "input channels and nInputPlane dont match");
  }

  long inputDepth  = input->size[2];
  long inputHeight  = input->size[3];
  long inputWidth   = input->size[4];
  long outputDepth  = (inputDepth  + 2*padT - (dilationT * (kT - 1) + 1)) / dT + 1;
  long outputHeight = (inputHeight + 2*padH - (dilationH * (kH - 1) + 1)) / dH + 1;
  long outputWidth  = (inputWidth  + 2*padW - (dilationW * (kW - 1) + 1)) / dW + 1;

  if (outputDepth < 1 || outputWidth < 1 || outputHeight < 1)
    THError("Given input size: (%dx%dx%dx%d). Calculated output size: (%dx%dx%dx%d). Output size is too small",
            nInputPlane,inputDepth,inputHeight,inputWidth,nOutputPlane,outputDepth,outputHeight,outputWidth);

  // Batch size + input planes
  long batchSize = input->size[0];

  // Resize output
  THCTensor_(resize5d)(state, output, batchSize, nOutputPlane, outputDepth, outputHeight, outputWidth);

  // Resize temporary columns
  THCTensor_(resize2d)(state, columns, nInputPlane*kT*kW*kH, outputDepth*outputHeight*outputWidth);

  // Define a buffer of ones, for bias accumulation
  // Note: this buffer can be shared with other modules, it only ever gets increased,
  // and always contains ones.
  if (ones->nDimension != 2 || ones->size[0]*ones->size[1]*ones->size[2] < outputDepth*outputHeight*outputWidth) {
    // Resize plane and fill with ones...
    THCTensor_(resize3d)(state, ones, outputDepth, outputHeight, outputWidth);
    THCTensor_(fill)(state, ones, ScalarConvert<int, real>::to(1));
  }

  // Helpers
  THCTensor  *input_n = THCTensor_(new)(state);
  THCTensor  *output_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per output:
    THCTensor_(select)(state, input_n, input, 0, elt);
    THCTensor_(select)(state, output_n, output, 0, elt);

    // Do Bias first:
    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m_ = nOutputPlane;
    long n_ = outputDepth * outputHeight * outputWidth;
    long k_ = 1;

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    if (bias) {
      #ifdef THC_REAL_IS_FLOAT
      THCudaBlas_Sgemm(
      #elif defined(THC_REAL_IS_HALF)
      THCudaBlas_Hgemm(
      #elif defined(THC_REAL_IS_DOUBLE)
      THCudaBlas_Dgemm(
      #endif
          state,
          't', 'n',
          n_, m_, k_,
          ScalarConvert<int, real>::to(1),
          THCTensor_(data)(state, ones), k_,
          THCTensor_(data)(state, bias), k_,
          ScalarConvert<int, real>::to(0),
          THCTensor_(data)(state, output_n), n_
      );
    } else {
      THCTensor_(zero)(state, output_n);
    }

    // Extract columns:
    vol2col(
      THCState_getCurrentStream(state),
      THCTensor_(data)(state, input_n),
      nInputPlane, inputDepth, inputHeight, inputWidth,
      kT, kH, kW, padT, padH, padW, dT, dH, dW,
      dilationT, dilationH, dilationW,
      THCTensor_(data)(state, columns)
    );

    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m = nOutputPlane;
    long n = columns->size[1];
    long k = nInputPlane*kT*kH*kW;

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    #ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemm(
    #elif defined(THC_REAL_IS_HALF)
    THCudaBlas_Hgemm(
    #elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemm(
    #endif
        state,
        'n', 'n',
        n, m, k,
        ScalarConvert<int, real>::to(1),
        THCTensor_(data)(state, columns), n,
        THCTensor_(data)(state, weight), k,
        ScalarConvert<int, real>::to(1),
        THCTensor_(data)(state, output_n), n
    );
  }

  // Free
  THCTensor_(free)(state, input_n);
  THCTensor_(free)(state, output_n);

  // Resize output
  if (batch == 0) {
    THCTensor_(resize4d)(state, output, nOutputPlane, outputDepth, outputHeight, outputWidth);
    THCTensor_(resize4d)(state, input, nInputPlane, inputDepth, inputHeight, inputWidth);
  }
}

void THNN_(VolumetricDilatedConvolution_updateGradInput)(
           THCState *state,
           THCTensor  *input,
           THCTensor  *gradOutput,
           THCTensor  *gradInput,
           THCTensor  *weight,
           THCTensor  *gradColumns,
           int kT, int kW, int kH,
           int dT, int dW, int dH,
           int padT, int padW, int padH,
           int dilationT, int dilationW, int dilationH) {

  THCUNN_assertSameGPU(state, 5, input, gradOutput, weight,
                       gradColumns, gradInput);
  THCUNN_argCheck(state, input->nDimension == 4 || input->nDimension == 5, 2, input,
                  "4D or 5D (batch mode) tensor expected for input, but got: %s");
  THCUNN_argCheck(state, gradOutput->nDimension == 4 || gradOutput->nDimension == 5, 3,
                  gradOutput,
                  "4D or 5D (batch mode) tensor expected for gradOutput, but got: %s");
  THCUNN_argCheck(state, weight->nDimension == 5, 4, weight,
                  "5D (nOutputPlane x nInputPlane x kT x kH x kW) tensor "
                  "expected for weight, but got: %s");
  THArgCheck(kT > 0 && kW > 0 && kH > 0, 8, "kernel size should be greater than zero");
  THArgCheck(dT > 0 && dW > 0 && dH > 0, 10, "stride should be greater than zero");

  // Params
  int nInputPlane = weight->size[1];
  int nOutputPlane = weight->size[0];

  int batch = 1;
  if (input->nDimension == 4) {
    // Force batch
    batch = 0;
    THCTensor_(resize5d)(state, input, 1, input->size[0], input->size[1], input->size[2], input->size[3]);
    THCTensor_(resize5d)(state, gradOutput, 1, gradOutput->size[0], gradOutput->size[1], gradOutput->size[2], gradOutput->size[3]);
  }

  long inputDepth  = input->size[2];
  long inputWidth   = input->size[4];
  long inputHeight  = input->size[3];
  long outputDepth  = (inputDepth + 2*padT - (dilationT * (kT - 1) + 1)) / dT + 1;
  long outputWidth  = (inputWidth + 2*padW - (dilationW * (kW - 1) + 1)) / dW + 1;
  long outputHeight = (inputHeight + 2*padH - (dilationH * (kH - 1) + 1)) / dH + 1;

  // Batch size + input planes
  long batchSize = input->size[0];

  // Resize output
  THCTensor_(resize5d)(state, gradInput, batchSize, nInputPlane, inputDepth, inputHeight, inputWidth);

  // Resize temporary columns
  THCTensor_(resize2d)(state, gradColumns, nInputPlane*kT*kW*kH, outputDepth*outputHeight*outputWidth);

  // Helpers
  THCTensor  *gradInput_n = THCTensor_(new)(state);
  THCTensor  *gradOutput_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per sample:
    THCTensor_(select)(state, gradInput_n, gradInput, 0, elt);
    THCTensor_(select)(state, gradOutput_n, gradOutput, 0, elt);

    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m = nInputPlane*kT*kW*kH;
    long n = gradColumns->size[1];
    long k = nOutputPlane;

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    #ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemm(
    #elif defined(THC_REAL_IS_HALF)
    THCudaBlas_Hgemm(
    #elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemm(
    #endif
        state,
        'n', 't',
        n, m, k,
        ScalarConvert<int, real>::to(1),
        THCTensor_(data)(state, gradOutput_n), n,
        THCTensor_(data)(state, weight), m,
        ScalarConvert<int, real>::to(0),
        THCTensor_(data)(state, gradColumns), n
    );

    // Unpack columns back into input:
    col2vol<real, accreal>(
      THCState_getCurrentStream(state),
      THCTensor_(data)(state, gradColumns),
      nInputPlane, inputDepth, inputHeight, inputWidth,
      kT, kH, kW, padT, padH, padW, dT, dH, dW,
      dilationT, dilationH, dilationW,
      THCTensor_(data)(state, gradInput_n)
    );
  }

  // Free
  THCTensor_(free)(state, gradInput_n);
  THCTensor_(free)(state, gradOutput_n);

  // Resize output
  if (batch == 0) {
    THCTensor_(resize4d)(state, gradOutput, nOutputPlane, outputDepth, outputHeight, outputWidth);
    THCTensor_(resize4d)(state, input, nInputPlane, inputDepth, inputHeight, inputWidth);
    THCTensor_(resize4d)(state, gradInput, nInputPlane, inputDepth, inputHeight, inputWidth);
  }
}

void THNN_(VolumetricDilatedConvolution_accGradParameters)(
           THCState *state,
           THCTensor  *input,
           THCTensor  *gradOutput,
           THCTensor  *gradWeight,
           THCTensor  *gradBias,
           THCTensor  *columns,
           THCTensor  *ones,
           int kT, int kW, int kH,
           int dT, int dW, int dH,
           int padT, int padW, int padH,
           int dilationT, int dilationW, int dilationH,
           real scale) {

  THCUNN_assertSameGPU(state, 5, input, gradOutput, gradWeight, columns, ones);
  if (gradBias) {
   THCUNN_assertSameGPU(state, 2, gradWeight, gradBias);
  }
  THCUNN_argCheck(state, input->nDimension == 4 || input->nDimension == 5, 2, input,
                  "4D or 5D (batch mode) tensor expected for input, but got: %s");
  THCUNN_argCheck(state, gradOutput->nDimension == 4 || gradOutput->nDimension == 5, 3,
                  gradOutput,
                  "4D or 5D (batch mode) tensor expected for gradOutput, but got: %s");
  THCUNN_argCheck(state, gradWeight->nDimension == 5, 4, gradWeight,
                  "5D (nOutputPlane x nInputPlane x kT x kH x kW) tensor "
                  "expected for gradWeight, but got: %s");
  THArgCheck(kT > 0 && kW > 0 && kH > 0, 8, "kernel size should be greater than zero");
  THArgCheck(dT > 0 && dW > 0 && dH > 0, 10, "stride should be greater than zero");

  // Params
  int nInputPlane = gradWeight->size[1];
  int nOutputPlane = gradWeight->size[0];

  int batch = 1;
  if (input->nDimension == 4) {
    // Force batch
    batch = 0;
    THCTensor_(resize5d)(state, input, 1, input->size[0], input->size[1], input->size[2], input->size[3]);
    THCTensor_(resize5d)(state, gradOutput, 1, gradOutput->size[0], gradOutput->size[1], gradOutput->size[2], gradOutput->size[3]);
  }

  long inputDepth  = input->size[2];
  long inputWidth   = input->size[4];
  long inputHeight  = input->size[3];
  long outputDepth  = (inputDepth + 2*padT - (dilationT * (kT - 1) + 1)) / dT + 1;
  long outputWidth  = (inputWidth + 2*padW - (dilationW * (kW - 1) + 1)) / dW + 1;
  long outputHeight = (inputHeight + 2*padH - (dilationH * (kH - 1) + 1)) / dH + 1;

  // Batch size + input planes
  long batchSize = input->size[0];

  // Define a buffer of ones, for bias accumulation
  if (ones->nDimension != 3 || ones->size[0]*ones->size[1]*ones->size[2] < outputDepth*outputHeight*outputWidth) {
    // Resize plane and fill with ones...
    THCTensor_(resize3d)(state, ones, outputDepth, outputHeight, outputWidth);
    THCTensor_(fill)(state, ones, ScalarConvert<int, real>::to(1));
  }

  // Resize temporary columns
  THCTensor_(resize2d)(state, columns, nInputPlane*kT*kW*kH, outputDepth*outputHeight*outputWidth);

  // Helpers
  THCTensor  *input_n = THCTensor_(new)(state);
  THCTensor  *gradOutput_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per output:
    THCTensor_(select)(state, input_n, input, 0, elt);
    THCTensor_(select)(state, gradOutput_n, gradOutput, 0, elt);

    // Extract columns:
    vol2col(
      THCState_getCurrentStream(state),
      THCTensor_(data)(state, input_n),
      nInputPlane, inputDepth, inputHeight, inputWidth, kT, kH, kW, padT, padH, padW, dT, dH, dW,
      dilationT, dilationH, dilationW,
      THCTensor_(data)(state, columns)
    );

    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m = nOutputPlane;
    long n = nInputPlane*kT*kW*kH;
    long k = columns->size[1];

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    #ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemm(
    #elif defined(THC_REAL_IS_HALF)
    THCudaBlas_Hgemm(
    #elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemm(
    #endif
        state,
        't', 'n',
        n, m, k,
        scale,
        THCTensor_(data)(state, columns), k,
        THCTensor_(data)(state, gradOutput_n), k,
        ScalarConvert<int, real>::to(1),
        THCTensor_(data)(state, gradWeight), n
    );

    // Do Bias:
    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m_ = nOutputPlane;
    long k_ = outputDepth * outputHeight * outputWidth;

    // Do GEMV (note: this is a bit confusing because gemv assumes column-major matrices)
    if (gradBias) {
      #if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
      #ifdef THC_REAL_IS_FLOAT
      THCudaBlas_Sgemv(
      #elif defined(THC_REAL_IS_DOUBLE)
      THCudaBlas_Dgemv(
      #endif
          state,
          't',
          k_, m_,
          scale,
          THCTensor_(data)(state, gradOutput_n), k_,
          THCTensor_(data)(state, ones), 1,
          ScalarConvert<int, real>::to(1),
          THCTensor_(data)(state, gradBias), 1
      );
      #endif
      #ifdef THC_REAL_IS_HALF
      THCudaBlas_Hgemm(
          state,
          't', 'n',
          m_, 1, k_,
          scale,
          THCTensor_(data)(state, gradOutput_n), k_,
          THCTensor_(data)(state, ones), k_,
          ScalarConvert<int, real>::to(1),
          THCTensor_(data)(state, gradBias), m_
      );
      #endif
    }
  }

  // Free
  THCTensor_(free)(state, input_n);
  THCTensor_(free)(state, gradOutput_n);

    // Resize output
  if (batch == 0) {
    THCTensor_(resize4d)(state, gradOutput, nOutputPlane, outputDepth, outputHeight, outputWidth);
    THCTensor_(resize4d)(state, input, nInputPlane, inputDepth, inputHeight, inputWidth);
  }
}

#endif
