
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/HardTanh.cu"
#else

#include "../common.h"

void THNN_(HardTanh_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           real min_val,
           real max_val,
           bool inplace)
{
  THCUNN_assertSameGPU(state, 2, input, output);
  if(inplace)
  {
    THCTensor_(set)(state, output, input);
    THC_pointwiseApply1(state, output, hardtanhupdateOutput_functor<real>(min_val, max_val));
  }
  else
  {
    THCTensor_(resizeAs)(state, output, input);
    THC_pointwiseApply2(state, output, input,
                               hardtanhupdateOutput_functor<real>(min_val, max_val));
  }
}

void THNN_(HardTanh_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           real min_val,
           real max_val,
           bool inplace)
{
  THCUNN_check_nElement(state, input, gradOutput);
  THCUNN_assertSameGPU(state, 3, input, gradOutput, gradInput);

  if (inplace)
  {
    THCTensor_(set)(state, gradInput, gradOutput);
    THC_pointwiseApply2(state, gradInput, input,
                                 hardtanhupdateGradInput_functor<real>(min_val, max_val));
  }
  else
  {
    THCTensor_(resizeAs)(state, gradInput, input);
    THC_pointwiseApply3(state, gradInput, input, gradOutput,
                                 hardtanhupdateGradInput_functor<real>(min_val, max_val));
  }
}

#endif
