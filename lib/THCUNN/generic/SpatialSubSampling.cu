
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SpatialSubSampling.cu"
#else

#include "../common.h"

void THNN_(SpatialSubSampling_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCTensor *weight,
           THCTensor *bias,
           int kW, int kH,
           int dW, int dH)
{
  real *weight_data = THCTensor_(data)(state, weight);
  real *bias_data = THCTensor_(data)(state, bias);
  real *output_data;
  real *input_data;

  int nInputPlane = THCTensor_(size)(state, weight, 0);

  THCUNN_assertSameGPU(state, 4, input, output, weight, bias);
  THArgCheck(input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch) tensor expected");

  if (input->nDimension == 3) {
    long nInputCols = input->size[2];
    long nInputRows = input->size[1];
    long nOutputCols = (nInputCols - kW) / dW + 1;
    long nOutputRows = (nInputRows - kH) / dH + 1;

    THArgCheck(input->size[0] == nInputPlane, 2, "invalid number of input planes");
    THArgCheck(nInputCols >= kW && nInputRows >= kH, 2, "input image smaller than kernel size");

    input = THCTensor_(newContiguous)(state, input);
    input_data = THCTensor_(data)(state, input);

    THCTensor_(resize3d)(state, output, nInputPlane, nOutputRows, nOutputCols);
    output_data = THCTensor_(data)(state, output);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane,yblocks);
    dim3 threads(32,8);

    // run subsample kernel
    subsample<real, accreal> <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (
      input_data, output_data, weight_data, bias_data,
      nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW);
    THCudaCheck(cudaGetLastError());
  } else {
    long nInputCols = input->size[3];
    long nInputRows = input->size[2];
    long nbatch = input->size[0];
    long nOutputCols = (nInputCols - kW) / dW + 1;
    long nOutputRows = (nInputRows - kH) / dH + 1;

    THArgCheck(input->size[1] == nInputPlane, 2, "invalid number of input planes");
    THArgCheck(nInputCols >= kW && nInputRows >= kH, 2, "input image smaller than kernel size");

    input = THCTensor_(newContiguous)(state, input);
    input_data = THCTensor_(data)(state, input);

    THCTensor_(resize4d)(state, output, nbatch, nInputPlane, nOutputRows, nOutputCols);
    output_data = THCTensor_(data)(state, output);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane*nbatch,yblocks);
    dim3 threads(32,8);

    // run subsample kernel
    subsample<real, accreal> <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (
      input_data, output_data, weight_data, bias_data,
      nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW);
    THCudaCheck(cudaGetLastError());
  }

  // clean
  THCTensor_(free)(state, input);

}

void THNN_(SpatialSubSampling_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *weight,
           int kW, int kH,
           int dW, int dH)
{
  THCUNN_assertSameGPU(state, 4, input, gradOutput, weight, gradInput);

  int nInputPlane = THCTensor_(size)(state, weight, 0);

  if (input->nDimension == 3) {
    long nInputCols = input->size[2];
    long nInputRows = input->size[1];

    real *weight_data = THCTensor_(data)(state, weight);
    real *gradOutput_data = THCTensor_(data)(state, gradOutput);
    real *gradInput_data;

    THCTensor_(resizeAs)(state, gradInput, input);
    THCTensor_(zero)(state, gradInput);
    gradInput_data = THCTensor_(data)(state, gradInput);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane,yblocks);
    dim3 threads(32,8);

    // run updateGradInput kernel
    if (kH <= dH && kW <= dW) {
      subgradinput <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (
        gradInput_data, gradOutput_data, weight_data,
        nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW);
    } else {
      subgradinputAtomic <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (
        gradInput_data, gradOutput_data, weight_data,
        nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW);
    }
    THCudaCheck(cudaGetLastError());
  } else {
    long nInputCols = input->size[3];
    long nInputRows = input->size[2];
    long nbatch = input->size[0];

    real *weight_data = THCTensor_(data)(state, weight);
    real *gradOutput_data = THCTensor_(data)(state, gradOutput);
    real *gradInput_data;

    THCTensor_(resizeAs)(state, gradInput, input);
    THCTensor_(zero)(state, gradInput);
    gradInput_data = THCTensor_(data)(state, gradInput);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane*nbatch,yblocks);
    dim3 threads(32,8);

    // run updateGradInput kernel
    if (kH <= dH && kW <= dW) {
      subgradinput <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (
        gradInput_data, gradOutput_data, weight_data,
        nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW);
    } else {
      subgradinputAtomic <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (
        gradInput_data, gradOutput_data, weight_data,
        nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW);
    }
    THCudaCheck(cudaGetLastError());
  }
}

void THNN_(SpatialSubSampling_accGradParameters)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradWeight,
           THCTensor *gradBias,
           int kW, int kH,
           int dW, int dH,
           float scale)
{
  THCUNN_assertSameGPU(state, 4, input, gradOutput, gradWeight, gradBias);

  int nInputPlane = THCTensor_(size)(state, gradWeight, 0);

  if (input->nDimension == 3) {
    long nInputCols = input->size[2];
    long nInputRows = input->size[1];

    real *gradWeight_data = THCTensor_(data)(state, gradWeight);
    real *gradBias_data = THCTensor_(data)(state, gradBias);
    real *gradOutput_data = THCTensor_(data)(state, gradOutput);
    real *input_data;

    input = THCTensor_(newContiguous)(state, input);
    input_data = THCTensor_(data)(state, input);

    // cuda blocks & threads:
    dim3 blocks(nInputPlane);
    dim3 threads(32,8);

    // run gradweight kernel
    subgradweight<real, accreal> <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (
      input_data, gradOutput_data, gradWeight_data, gradBias_data,
      nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW, scale);
    THCudaCheck(cudaGetLastError());
  } else {
    long nInputCols = input->size[3];
    long nInputRows = input->size[2];
    long nbatch = input->size[0];

    real *gradWeight_data = THCTensor_(data)(state, gradWeight);
    real *gradBias_data = THCTensor_(data)(state, gradBias);
    real *gradOutput_data = THCTensor_(data)(state, gradOutput);
    real *input_data;

    input = THCTensor_(newContiguous)(state, input);
    input_data = THCTensor_(data)(state, input);

    // cuda blocks & threads:
    dim3 blocks(nInputPlane);
    dim3 threads(32,8);

    // run gradweight kernel
    long sl;
    for (sl=0; sl<nbatch; sl++) {
      subgradweight<real, accreal> <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (
        input_data + sl*input->stride[0],
        gradOutput_data + sl*gradOutput->stride[0],
        gradWeight_data, gradBias_data,
        nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW, scale);
    }
    THCudaCheck(cudaGetLastError());
  }

  // clean
  THCTensor_(free)(state, input);

}

#endif
